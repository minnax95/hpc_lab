%%writefile VectorVectorMultiply.cu

#include<stdio.h>
#include<hip/hip_runtime.h>
#define size 4 /* Number of rows of first matrix */


_global_ void matproduct(int *l,int *m, int *n)
{
    int k;
 
  for(k=0;k<size;k++)
    {
      n[k]=l[k]*m[k];
    }
}

int main()
{
    int a[size];
    int b[size];
    int c[size];
    int *d,*e,*f;
    int i;

    printf("\nElements of first vector of size 4\n");
    for(i=0;i<size;i++)
    {
        a[i] = i+1;
        printf("%d ",a[i]);
    
            
    }
  printf("\n");
    printf("\nElements of second vector of size 4\n");
        for(i=0;i<size;i++)
        {
            b[i] = i+2;
            printf("%d ",b[i]);
            
        }
 printf("\n");

    hipMalloc((void **)&d,size*sizeof(int));
    hipMalloc((void **)&e,size*sizeof(int));
    hipMalloc((void **)&f,size*sizeof(int));

 hipMemcpy(d,a,size*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,size*sizeof(int),hipMemcpyHostToDevice);

  dim3 grid(1,size);

    matproduct<<<grid,1>>>(d,e,f);

 hipMemcpy(c,f,size*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nProduct of two vectors:\n ");
    for(i=0;i<size;i++)
    {
       printf("%d ",c[i]);
   
        
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}